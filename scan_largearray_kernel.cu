#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
// MP4.2 - You can use any other block size you wish.
#define BLOCK_SIZE 256

// MP4.2 - Host Helper Functions (allocate your own data structure...)


// MP4.2 - Device Functions


// MP4.2 - Kernel Functions


// **===-------- MP4.2 - Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{



}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
